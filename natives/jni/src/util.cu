/*
*   This program is free software: you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation, version 2.1
*
*   This program is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License, version 2.1, for more details.
*
*   You should have received a copy of the GNU General Public License
*
*/

#include "bb/segments/kv/bb_segsort.hpp"
#include "bb/segments/k/bb_segsort.hpp"
#include "bb/matrix/kv/bb_segsort.hpp"
#include "bb/matrix/k/bb_segsort.hpp"

#include "util.h"

template <class K>
int sort_matrix(K* key, int rows, int cols, const bb::k::SortContext<K>* context) {
  hipError_t err;
  K    *key_d;

  err = hipMalloc((void**)&key_d, sizeof(K) * rows * cols);
  CUDA_CHECK(err, "segments: alloc key_d");

  err = hipMemcpy(key_d, key, sizeof(K) * rows * cols, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to key_d");

  bb::matrix::k::bb_segsort(key_d, rows, cols, context);

  err = hipFree(key_d);
  CUDA_CHECK(err, "segments: free key_d");

  return 0;
}

template <class K, class V>
int sort_matrix(K* key, V* val, int rows, int cols, const bb::kv::SortContext<K, V>* context) {
  hipError_t err;
  K    *key_d;
  V    *val_d;

  err = hipMalloc((void**)&key_d, sizeof(K) * rows * cols);
  CUDA_CHECK(err, "segments: alloc key_d");
  err = hipMalloc((void**)&val_d, sizeof(V) * rows * cols);
  CUDA_CHECK(err, "segments: alloc val_d");

  err = hipMemcpy(key_d, key, sizeof(K) * rows * cols, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to key_d");
  err = hipMemcpy(val_d, val, sizeof(V) * rows * cols, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to val_d");

  bb::matrix::kv::bb_segsort(key_d, val_d, rows, cols, context);

  err = hipFree(key_d);
  CUDA_CHECK(err, "segments: free key_d");
  err = hipFree(val_d);
  CUDA_CHECK(err, "segments: free val_d");

  return 0;
}

template <class K>
int sort_matrix(K* key, int rows, int cols) {
  bb::k::SortContext<K> context_k(rows * cols, rows);
  return sort_matrix(key, rows, cols, &context_k);
}

template <class K, class V>
int sort_matrix(K* key, V* val, int rows, int cols) {
  bb::kv::SortContext<K, V> context_kv(rows * cols, rows);
  return sort_matrix(key, val, rows, cols, &context_kv);
}


template <class K>
int sort_segments(K* key, int* seg, int n, int length, const bb::k::SortContext<K>* context) {
  hipError_t err;
  K    *key_d;
  int  *seg_d;

  err = hipMalloc((void**)&key_d, sizeof(K)*n);
  CUDA_CHECK(err, "segments: alloc key_d");
  err = hipMalloc((void**)&seg_d, sizeof(int)*length);
  CUDA_CHECK(err, "segments: alloc seg_d");

  err = hipMemcpy(key_d, key, sizeof(K)*n, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to key_d");
  err = hipMemcpy(seg_d, seg, sizeof(int)*length, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to seg_d");

  bb::segments::k::bb_segsort(key_d, n, seg_d, length, context);

  err = hipFree(key_d);
  CUDA_CHECK(err, "segments: free key_d");
  err = hipFree(seg_d);
  CUDA_CHECK(err, "segments: free seg_d");

  return 0;
}

template <class K, class V>
int sort_segments(K* key, V* val, int* seg, int n, int length, const bb::kv::SortContext<K, V>* context) {
  hipError_t err;
  K    *key_d;
  V    *val_d;
  int  *seg_d;

  err = hipMalloc((void**)&key_d, sizeof(K)*n);
  CUDA_CHECK(err, "segments: alloc key_d");
  err = hipMalloc((void**)&val_d, sizeof(V)*n);
  CUDA_CHECK(err, "segments: alloc val_d");
  err = hipMalloc((void**)&seg_d, sizeof(int)*length);
  CUDA_CHECK(err, "segments: alloc seg_d");

  err = hipMemcpy(key_d, key, sizeof(K)*n, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to key_d");
  err = hipMemcpy(val_d, val, sizeof(V)*n, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to val_d");
  err = hipMemcpy(seg_d, seg, sizeof(int)*length, hipMemcpyHostToDevice);
  CUDA_CHECK(err, "segments: copy to seg_d");

  bb::segments::kv::bb_segsort(key_d, val_d, n, seg_d, length, context);

  err = hipFree(key_d);
  CUDA_CHECK(err, "segments: free key_d");
  err = hipFree(val_d);
  CUDA_CHECK(err, "segments: free val_d");
  err = hipFree(seg_d);
  CUDA_CHECK(err, "segments: free seg_d");

  return 0;
}

template <class K>
int sort_segments(K* key, int* seg, int n, int length) {
  bb::k::SortContext<K> context_k(n, length);
  return sort_segments(key, seg, n, length, &context_k);
}

template <class K, class V>
int sort_segments(K* key, V* val, int* seg, int n, int length) {
  bb::kv::SortContext<K, V> context_kv(n, length);
  return sort_segments(key, val, seg, n, length, &context_kv);
}

///////////////// sort keys matrix /////////////////////////////////////////////////
int sort_matrix_unsigned_char(unsigned char *key, int rows, int cols) {
  return sort_matrix<unsigned char>(key, rows, cols);
}

int sort_matrix_char(char *key, int rows, int cols) {
  return sort_matrix<char>(key, rows, cols);
}

int sort_matrix_short(short *key, int rows, int cols) {
  return sort_matrix<short>(key, rows, cols);
}

int sort_matrix_int(int *key, int rows, int cols) {
  return sort_matrix<int>(key, rows, cols);
}

int sort_matrix_long_long_int(long long int *key, int rows, int cols) {
  return sort_matrix<long long int>(key, rows, cols);
}

int sort_matrix_float(float *key, int rows, int cols) {
  return sort_matrix<float>(key, rows, cols);
}

int sort_matrix_double(double *key, int rows, int cols) {
  return sort_matrix<double>(key, rows, cols);
}

///////////////// sort key-value pairs matrix ///////////////////////////////////////////
int sort_matrix_unsigned_char_unsigned_char(unsigned char *key, unsigned char *val, int rows, int cols) {
  return sort_matrix<unsigned char, unsigned char>(key, val, rows, cols);
}

int sort_matrix_unsigned_char_char(unsigned char *key, char *val, int rows, int cols) {
  return sort_matrix<unsigned char, char>(key, val, rows, cols);
}

int sort_matrix_unsigned_char_short(unsigned char *key, short *val, int rows, int cols) {
  return sort_matrix<unsigned char, short>(key, val, rows, cols);
}

int sort_matrix_unsigned_char_int(unsigned char *key, int *val, int rows, int cols) {
  return sort_matrix<unsigned char, int>(key, val, rows, cols);
}

int sort_matrix_unsigned_char_long_long_int(unsigned char *key, long long int *val, int rows, int cols) {
  return sort_matrix<unsigned char, long long int>(key, val, rows, cols);
}

int sort_matrix_unsigned_char_float(unsigned char *key, float *val, int rows, int cols) {
  return sort_matrix<unsigned char, float>(key, val, rows, cols);
}

int sort_matrix_unsigned_char_double(unsigned char *key, double *val, int rows, int cols) {
  return sort_matrix<unsigned char, double>(key, val, rows, cols);
}

int sort_matrix_char_unsigned_char(char *key, unsigned char *val, int rows, int cols) {
  return sort_matrix<char, unsigned char>(key, val, rows, cols);
}

int sort_matrix_char_char(char *key, char *val, int rows, int cols) {
  return sort_matrix<char, char>(key, val, rows, cols);
}

int sort_matrix_char_short(char *key, short *val, int rows, int cols) {
  return sort_matrix<char, short>(key, val, rows, cols);
}

int sort_matrix_char_int(char *key, int *val, int rows, int cols) {
  return sort_matrix<char, int>(key, val, rows, cols);
}

int sort_matrix_char_long_long_int(char *key, long long int *val, int rows, int cols) {
  return sort_matrix<char, long long int>(key, val, rows, cols);
}

int sort_matrix_char_float(char *key, float *val, int rows, int cols) {
  return sort_matrix<char, float>(key, val, rows, cols);
}

int sort_matrix_char_double(char *key, double *val, int rows, int cols) {
  return sort_matrix<char, double>(key, val, rows, cols);
}

int sort_matrix_short_unsigned_char(short *key, unsigned char *val, int rows, int cols) {
  return sort_matrix<short, unsigned char>(key, val, rows, cols);
}

int sort_matrix_short_char(short *key, char *val, int rows, int cols) {
  return sort_matrix<short, char>(key, val, rows, cols);
}

int sort_matrix_short_short(short *key, short *val, int rows, int cols) {
  return sort_matrix<short, short>(key, val, rows, cols);
}

int sort_matrix_short_int(short *key, int *val, int rows, int cols) {
  return sort_matrix<short, int>(key, val, rows, cols);
}

int sort_matrix_short_long_long_int(short *key, long long int *val, int rows, int cols) {
  return sort_matrix<short, long long int>(key, val, rows, cols);
}

int sort_matrix_short_float(short *key, float *val, int rows, int cols) {
  return sort_matrix<short, float>(key, val, rows, cols);
}

int sort_matrix_short_double(short *key, double *val, int rows, int cols) {
  return sort_matrix<short, double>(key, val, rows, cols);
}

int sort_matrix_int_unsigned_char(int *key, unsigned char *val, int rows, int cols) {
  return sort_matrix<int, unsigned char>(key, val, rows, cols);
}

int sort_matrix_int_char(int *key, char *val, int rows, int cols) {
  return sort_matrix<int, char>(key, val, rows, cols);
}

int sort_matrix_int_short(int *key, short *val, int rows, int cols) {
  return sort_matrix<int, short>(key, val, rows, cols);
}

int sort_matrix_int_int(int *key, int *val, int rows, int cols) {
  return sort_matrix<int, int>(key, val, rows, cols);
}

int sort_matrix_int_long_long_int(int *key, long long int *val, int rows, int cols) {
  return sort_matrix<int, long long int>(key, val, rows, cols);
}

int sort_matrix_int_float(int *key, float *val, int rows, int cols) {
  return sort_matrix<int, float>(key, val, rows, cols);
}

int sort_matrix_int_double(int *key, double *val, int rows, int cols) {
  return sort_matrix<int, double>(key, val, rows, cols);
}

int sort_matrix_long_long_int_unsigned_char(long long int *key, unsigned char *val, int rows, int cols) {
  return sort_matrix<long long int, unsigned char>(key, val, rows, cols);
}

int sort_matrix_long_long_int_char(long long int *key, char *val, int rows, int cols) {
  return sort_matrix<long long int, char>(key, val, rows, cols);
}

int sort_matrix_long_long_int_short(long long int *key, short *val, int rows, int cols) {
  return sort_matrix<long long int, short>(key, val, rows, cols);
}

int sort_matrix_long_long_int_int(long long int *key, int *val, int rows, int cols) {
  return sort_matrix<long long int, int>(key, val, rows, cols);
}

int sort_matrix_long_long_int_long_long_int(long long int *key, long long int *val, int rows, int cols) {
  return sort_matrix<long long int, long long int>(key, val, rows, cols);
}

int sort_matrix_long_long_int_float(long long int *key, float *val, int rows, int cols) {
  return sort_matrix<long long int, float>(key, val, rows, cols);
}

int sort_matrix_long_long_int_double(long long int *key, double *val, int rows, int cols) {
  return sort_matrix<long long int, double>(key, val, rows, cols);
}

int sort_matrix_float_unsigned_char(float *key, unsigned char *val, int rows, int cols) {
  return sort_matrix<float, unsigned char>(key, val, rows, cols);
}

int sort_matrix_float_char(float *key, char *val, int rows, int cols) {
  return sort_matrix<float, char>(key, val, rows, cols);
}

int sort_matrix_float_short(float *key, short *val, int rows, int cols) {
  return sort_matrix<float, short>(key, val, rows, cols);
}

int sort_matrix_float_int(float *key, int *val, int rows, int cols) {
  return sort_matrix<float, int>(key, val, rows, cols);
}

int sort_matrix_float_long_long_int(float *key, long long int *val, int rows, int cols) {
  return sort_matrix<float, long long int>(key, val, rows, cols);
}

int sort_matrix_float_float(float *key, float *val, int rows, int cols) {
  return sort_matrix<float, float>(key, val, rows, cols);
}

int sort_matrix_float_double(float *key, double *val, int rows, int cols) {
  return sort_matrix<float, double>(key, val, rows, cols);
}

int sort_matrix_double_unsigned_char(double *key, unsigned char *val, int rows, int cols) {
  return sort_matrix<double, unsigned char>(key, val, rows, cols);
}

int sort_matrix_double_char(double *key, char *val, int rows, int cols) {
  return sort_matrix<double, char>(key, val, rows, cols);
}

int sort_matrix_double_short(double *key, short *val, int rows, int cols) {
  return sort_matrix<double, short>(key, val, rows, cols);
}

int sort_matrix_double_int(double *key, int *val, int rows, int cols) {
  return sort_matrix<double, int>(key, val, rows, cols);
}

int sort_matrix_double_long_long_int(double *key, long long int *val, int rows, int cols) {
  return sort_matrix<double, long long int>(key, val, rows, cols);
}

int sort_matrix_double_float(double *key, float *val, int rows, int cols) {
  return sort_matrix<double, float>(key, val, rows, cols);
}

int sort_matrix_double_double(double *key, double *val, int rows, int cols) {
  return sort_matrix<double, double>(key, val, rows, cols);
}

///////////////// sort keys by segments /////////////////////////////////////////////////
int sort_segments_unsigned_char(unsigned char *key, int* seg, int n, int length) {
  return sort_segments<unsigned char>(key, seg, n, length);
}

int sort_segments_char(char *key, int* seg, int n, int length) {
  return sort_segments<char>(key, seg, n, length);
}

int sort_segments_short(short *key, int* seg, int n, int length) {
  return sort_segments<short>(key, seg, n, length);
}

int sort_segments_int(int *key, int* seg, int n, int length) {
  return sort_segments<int>(key, seg, n, length);
}

int sort_segments_long_long_int(long long int *key, int* seg, int n, int length) {
  return sort_segments<long long int>(key, seg, n, length);
}

int sort_segments_float(float *key, int* seg, int n, int length) {
  return sort_segments<float>(key, seg, n, length);
}

int sort_segments_double(double *key, int* seg, int n, int length) {
  return sort_segments<double>(key, seg, n, length);
}

///////////////// sort key-value pairs by segments ///////////////////////////////////////////
int sort_segments_unsigned_char_unsigned_char(unsigned char *key, unsigned char *val, int *seg, int n, int length) {
  return sort_segments<unsigned char, unsigned char>(key, val, seg, n, length);
}

int sort_segments_unsigned_char_char(unsigned char *key, char *val, int *seg, int n, int length) {
  return sort_segments<unsigned char, char>(key, val, seg, n, length);
}

int sort_segments_unsigned_char_short(unsigned char *key, short *val, int *seg, int n, int length) {
  return sort_segments<unsigned char, short>(key, val, seg, n, length);
}

int sort_segments_unsigned_char_int(unsigned char *key, int *val, int *seg, int n, int length) {
  return sort_segments<unsigned char, int>(key, val, seg, n, length);
}

int sort_segments_unsigned_char_long_long_int(unsigned char *key, long long int *val, int *seg, int n, int length) {
  return sort_segments<unsigned char, long long int>(key, val, seg, n, length);
}

int sort_segments_unsigned_char_float(unsigned char *key, float *val, int *seg, int n, int length) {
  return sort_segments<unsigned char, float>(key, val, seg, n, length);
}

int sort_segments_unsigned_char_double(unsigned char *key, double *val, int *seg, int n, int length) {
  return sort_segments<unsigned char, double>(key, val, seg, n, length);
}

int sort_segments_char_unsigned_char(char *key, unsigned char *val, int *seg, int n, int length) {
  return sort_segments<char, unsigned char>(key, val, seg, n, length);
}

int sort_segments_char_char(char *key, char *val, int *seg, int n, int length) {
  return sort_segments<char, char>(key, val, seg, n, length);
}

int sort_segments_char_short(char *key, short *val, int *seg, int n, int length) {
  return sort_segments<char, short>(key, val, seg, n, length);
}

int sort_segments_char_int(char *key, int *val, int *seg, int n, int length) {
  return sort_segments<char, int>(key, val, seg, n, length);
}

int sort_segments_char_long_long_int(char *key, long long int *val, int *seg, int n, int length) {
  return sort_segments<char, long long int>(key, val, seg, n, length);
}

int sort_segments_char_float(char *key, float *val, int *seg, int n, int length) {
  return sort_segments<char, float>(key, val, seg, n, length);
}

int sort_segments_char_double(char *key, double *val, int *seg, int n, int length) {
  return sort_segments<char, double>(key, val, seg, n, length);
}

int sort_segments_short_unsigned_char(short *key, unsigned char *val, int *seg, int n, int length) {
  return sort_segments<short, unsigned char>(key, val, seg, n, length);
}

int sort_segments_short_char(short *key, char *val, int *seg, int n, int length) {
  return sort_segments<short, char>(key, val, seg, n, length);
}

int sort_segments_short_short(short *key, short *val, int *seg, int n, int length) {
  return sort_segments<short, short>(key, val, seg, n, length);
}

int sort_segments_short_int(short *key, int *val, int *seg, int n, int length) {
  return sort_segments<short, int>(key, val, seg, n, length);
}

int sort_segments_short_long_long_int(short *key, long long int *val, int *seg, int n, int length) {
  return sort_segments<short, long long int>(key, val, seg, n, length);
}

int sort_segments_short_float(short *key, float *val, int *seg, int n, int length) {
  return sort_segments<short, float>(key, val, seg, n, length);
}

int sort_segments_short_double(short *key, double *val, int *seg, int n, int length) {
  return sort_segments<short, double>(key, val, seg, n, length);
}

int sort_segments_int_unsigned_char(int *key, unsigned char *val, int *seg, int n, int length) {
  return sort_segments<int, unsigned char>(key, val, seg, n, length);
}

int sort_segments_int_char(int *key, char *val, int *seg, int n, int length) {
  return sort_segments<int, char>(key, val, seg, n, length);
}

int sort_segments_int_short(int *key, short *val, int *seg, int n, int length) {
  return sort_segments<int, short>(key, val, seg, n, length);
}

int sort_segments_int_int(int *key, int *val, int *seg, int n, int length) {
  return sort_segments<int, int>(key, val, seg, n, length);
}

int sort_segments_int_long_long_int(int *key, long long int *val, int *seg, int n, int length) {
  return sort_segments<int, long long int>(key, val, seg, n, length);
}

int sort_segments_int_float(int *key, float *val, int *seg, int n, int length) {
  return sort_segments<int, float>(key, val, seg, n, length);
}

int sort_segments_int_double(int *key, double *val, int *seg, int n, int length) {
  return sort_segments<int, double>(key, val, seg, n, length);
}

int sort_segments_long_long_int_unsigned_char(long long int *key, unsigned char *val, int *seg, int n, int length) {
  return sort_segments<long long int, unsigned char>(key, val, seg, n, length);
}

int sort_segments_long_long_int_char(long long int *key, char *val, int *seg, int n, int length) {
  return sort_segments<long long int, char>(key, val, seg, n, length);
}

int sort_segments_long_long_int_short(long long int *key, short *val, int *seg, int n, int length) {
  return sort_segments<long long int, short>(key, val, seg, n, length);
}

int sort_segments_long_long_int_int(long long int *key, int *val, int *seg, int n, int length) {
  return sort_segments<long long int, int>(key, val, seg, n, length);
}

int sort_segments_long_long_int_long_long_int(long long int *key, long long int *val, int *seg, int n, int length) {
  return sort_segments<long long int, long long int>(key, val, seg, n, length);
}

int sort_segments_long_long_int_float(long long int *key, float *val, int *seg, int n, int length) {
  return sort_segments<long long int, float>(key, val, seg, n, length);
}

int sort_segments_long_long_int_double(long long int *key, double *val, int *seg, int n, int length) {
  return sort_segments<long long int, double>(key, val, seg, n, length);
}

int sort_segments_float_unsigned_char(float *key, unsigned char *val, int *seg, int n, int length) {
  return sort_segments<float, unsigned char>(key, val, seg, n, length);
}

int sort_segments_float_char(float *key, char *val, int *seg, int n, int length) {
  return sort_segments<float, char>(key, val, seg, n, length);
}

int sort_segments_float_short(float *key, short *val, int *seg, int n, int length) {
  return sort_segments<float, short>(key, val, seg, n, length);
}

int sort_segments_float_int(float *key, int *val, int *seg, int n, int length) {
  return sort_segments<float, int>(key, val, seg, n, length);
}

int sort_segments_float_long_long_int(float *key, long long int *val, int *seg, int n, int length) {
  return sort_segments<float, long long int>(key, val, seg, n, length);
}

int sort_segments_float_float(float *key, float *val, int *seg, int n, int length) {
  return sort_segments<float, float>(key, val, seg, n, length);
}

int sort_segments_float_double(float *key, double *val, int *seg, int n, int length) {
  return sort_segments<float, double>(key, val, seg, n, length);
}

int sort_segments_double_unsigned_char(double *key, unsigned char *val, int *seg, int n, int length) {
  return sort_segments<double, unsigned char>(key, val, seg, n, length);
}

int sort_segments_double_char(double *key, char *val, int *seg, int n, int length) {
  return sort_segments<double, char>(key, val, seg, n, length);
}

int sort_segments_double_short(double *key, short *val, int *seg, int n, int length) {
  return sort_segments<double, short>(key, val, seg, n, length);
}

int sort_segments_double_int(double *key, int *val, int *seg, int n, int length) {
  return sort_segments<double, int>(key, val, seg, n, length);
}

int sort_segments_double_long_long_int(double *key, long long int *val, int *seg, int n, int length) {
  return sort_segments<double, long long int>(key, val, seg, n, length);
}

int sort_segments_double_float(double *key, float *val, int *seg, int n, int length) {
  return sort_segments<double, float>(key, val, seg, n, length);
}

int sort_segments_double_double(double *key, double *val, int *seg, int n, int length) {
  return sort_segments<double, double>(key, val, seg, n, length);
}
